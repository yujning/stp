#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include "eigen3/Eigen/Dense"
#include "eigen3/Eigen/Sparse"

#include <excute_cuda.hpp>


uint64_t Total_Thread = 0; // total supported threads


extern "C"
//get total thread number
void Get_Total_Thread_Num(void)
{
    int deviceCount;
    //get the number of CUDA devices
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
    {
        std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(error) << std::endl;
        return;
    }

    for (int i = 0; i < deviceCount; ++i) 
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        //std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        //std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        //std::cout << "Max blocks per grid: " << (int64_t)deviceProp.maxGridSize[0] * deviceProp.maxGridSize[1] * deviceProp.maxGridSize[2] << std::endl;
        //std::cout << "  X : " << deviceProp.maxGridSize[0] << std::endl;
        //std::cout << "  Y : " << deviceProp.maxGridSize[1] << std::endl;
        //std::cout << "  Z : " << deviceProp.maxGridSize[2] << std::endl;
        //std::cout << "Total max threads: " << deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount << std::endl;
        //std::cout << std::endl;
        Total_Thread = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount;
    }    
}

extern "C"
CUDA_DATA  Memcpy_To_Device(std::vector<stp_data>& A)
{
    CUDA_DATA C;
    C._row = A[0];
    C._col = A.size() - 1;

    //compute space 
    size_t size_A = (A.size() - 1) * sizeof(stp_data);

    //allocate memory
    hipError_t  errC = hipMalloc((void **)&C.d_Vec, size_A);
    if (errC != hipSuccess) 
    {
        std::cerr << "Error allocating memory for In_KR_Matrix d_C: " << hipGetErrorString(errC) << std::endl;
    }

    //Copy parameters
    hipMemcpy(C.d_Vec, A.data() + 1, size_A, hipMemcpyHostToDevice);

    return C;
}


extern "C"
//release GPU memory
bool Free_Device_Memory(CUDA_DATA& C)
{
    hipError_t err = hipFree(C.d_Vec);
    if (err != hipSuccess) 
    {
        std::cerr << "Error freeing memory for In_KR_Matrix d_C: " << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}


extern "C"
std::vector<stp_data> Memcpy_To_Host(CUDA_DATA& C)
{
    std::vector<stp_data> A(C._col + 1);
    A[0] = C._row;
    hipMemcpy(A.data() + 1, C.d_Vec, (C._col) * sizeof(stp_data), hipMemcpyDeviceToHost);
    hipFree(C.d_Vec);

    return A;
}

//In_KR_Matrix_Kernel     
__global__ void In_KR_Matrix_Kernel(int32_t sub_dim, int32_t idx_offset, stp_data *A, stp_data A_row, stp_data A_val_len, stp_data *C, stp_data C_val_len)
{
    stp_data ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    stp_data idx = ix + idx_offset;

    stp_data x_code = idx / A_val_len;

    stp_data y_code = idx % A_val_len;

    //boundary check  
    if(idx < C_val_len)
    {    
        //XP+Y
        C[idx] = x_code * A_row + A[y_code]; 
    }
}


extern "C"
CUDA_DATA my_cuda_In_KR_Matrix(int32_t dim, CUDA_DATA& A)
{
    //Get the dimensions of matrix A
    stp_data A_row = A._row;
    stp_data A_col = A._col;

    //Calculate the size of result matrix
    stp_data C_len = dim * A_col;

    CUDA_DATA C; 
    C._row = A_row * dim;
    C._col = A_col * dim;

    //Assign the number of rows of result matrix
    stp_data *d_C;

    //compute space 
    size_t size_C = C_len * sizeof(stp_data);

    //allocate memory
    hipError_t errC;

    errC = hipMalloc((void **)&d_C, size_C);
    if (errC != hipSuccess) 
    {
        std::cerr << "Error allocating memory for In_KR_Matrix d_C: " << hipGetErrorString(errC) << std::endl;
    }

    //Calculate the block size (maximum 1024)
    dim3 threadsPerBlock(1024, 1);

    //Can be done in one go (with each element of C as a thread)
    if((C_len) <=Total_Thread)
    {
        //Calculate the grid size (for large scale)
        dim3 numBlocks0(( C_len + 1024 -1 ) / 1024, 1);

        //Launch GPU
        In_KR_Matrix_Kernel<<<numBlocks0, threadsPerBlock>>>(dim, 0, A.d_Vec, A_row, A_col , d_C, C_len);
        hipDeviceSynchronize(); //Wait for the kernel to complete
    }
    //Divide into blocks (by column)
    else
    {
        int32_t remain_num = C_len; //remaining unassigned columns in C
        int32_t idx_offset = 0;  //Thread offset

        while(remain_num)
        {
            //the last  
            if(remain_num <= Total_Thread)
            {
                //Calculate the grid size (for large scale)
                dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                In_KR_Matrix_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, A.d_Vec, A_row, A_col, d_C, C_len); 
                hipDeviceSynchronize(); //Wait for the kernel to complete
                //Calculate the thread offset
                idx_offset += remain_num;
                remain_num = 0; //exit the loop
            }
            // Total thread
            else
            {
                //Calculate the grid size (for large scale)
                dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                In_KR_Matrix_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, A.d_Vec, A_row, A_col, d_C, C_len);  
                hipDeviceSynchronize(); //Wait for the kernel to complete

                //Calculate thread offset
                idx_offset += Total_Thread;
                remain_num -= Total_Thread; //exit the loop                
            }
        }     

    }

    //Free resources
    // if(A_col>20)
    // {
    //     hipFree(A.d_Vec);
    // }
    hipFree(A.d_Vec);
    C.d_Vec = d_C;

    return C;
}





//Matrix_KR_In_Kernel
__global__ void Matrix_KR_In_Kernel(int32_t dim, int32_t idx_offset, stp_data *A, stp_data A_val_len, stp_data *C, stp_data C_val_len)
{
    stp_data ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    stp_data idx = ix + idx_offset;

    //calculate x_code
    stp_data x_code = idx / dim;

    //calculate y_code
    stp_data y_code = idx % dim;

    //boundary check
    if(idx < C_val_len)
    {    
        //xp+y
        C[idx] = A[x_code] * dim + y_code;
    }
    
}

extern "C"
//Matrix_KR_In
CUDA_DATA my_cuda_Matrix_KR_In(int32_t dim,  CUDA_DATA& A)
{
    //get dimensions of matrix A
    stp_data A_row = A._row;
    stp_data A_col = A._col;

    //calculate size of result matrix
    stp_data C_len = A_col * dim;

    CUDA_DATA C;
    C._row = A_row * dim;
    C._col = A_col * dim;


    stp_data *d_C;
    //compute space
    size_t size_C = C_len * sizeof(stp_data);

    //allocate memory
    hipError_t errC;

    errC = hipMalloc((void **)&d_C, size_C);
    if (errC != hipSuccess) 
    {
        std::cerr << "Error allocating memory for Matrix_KR_In d_C: " << hipGetErrorString(errC) << std::endl;
    }

    //calculate block size (maximum 1024)
    dim3 threadsPerBlock(1024, 1);

    //can be done in one go (with each element of C as a thread)
    if((C_len - 1) <= Total_Thread)
    {
        //calculate grid size (for large scale)
        dim3 numBlocks(( C_len + 1024 -1 ) / 1024, 1);

        Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, 0, A.d_Vec, A_col, d_C, C_len); 
        hipDeviceSynchronize(); //wait for the kernel to complete
    }
    //divide into blocks (by column)
    else
    {
        int32_t remain_num = C_len; //C remaining unassigned columns
        int32_t idx_offset = 0;  //thread offset

        while(remain_num)
        {
            //last time  quantity remain_num
            if(remain_num <= Total_Thread)
            {
                //calculate grid size (for large scale) 
                dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, A.d_Vec, A_col, d_C, C_len);  
                hipDeviceSynchronize(); //wait for the kernel to complete
                //calculate thread offset
                idx_offset += remain_num;
                remain_num = 0; //exit the loop
            }
            //total thread
            else
            {
                //calculate grid size (for large scale)
                dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                Matrix_KR_In_Kernel<<<numBlocks, threadsPerBlock>>>(dim, idx_offset, A.d_Vec , A_col, d_C, C_len);  
                hipDeviceSynchronize(); //wait for the kernel to complete

                //calculate thread offset
                idx_offset += Total_Thread;
                remain_num -= Total_Thread; //exit the loop                
            }
        }
    }

    //free resources
    // if(A_col>20)
    // {
    //     hipFree(A.d_Vec);
    // }
    hipFree(A.d_Vec);
    
    C.d_Vec = d_C;

    return C;
}         



//Matrix_Multipiy_Kernel
__global__ void Matrix_Multipiy_Kernel(int32_t idx_offset, stp_data *A, int32_t A_val_len, stp_data *B, int32_t B_val_len, stp_data *C, int32_t C_val_len, int32_t t)
{
    int32_t ix = blockIdx.x * 1024 + threadIdx.x;
    //index
    int32_t idx = ix + idx_offset;

    //calculate x_code
    int32_t x_code = idx % t ;

    //calculate y_code
    int32_t y_code = idx / t;

    //boundary check
    if(idx < C_val_len )
    {
        //result
        C[idx] = A[ B[ y_code ] * t + x_code];
    }
}



extern "C"
//my_semi_tensor_product 
CUDA_DATA my_cuda_semi_tensor_product(CUDA_DATA& A, CUDA_DATA& B)
{
    //get dimensions of matrix A and B
    int32_t A_row = A._row;
    int32_t A_col = A._col;
    int32_t B_row = B._row;
    int32_t B_col = B._col;

    if(A_col % B_row == 0)
    {
        CUDA_DATA C;
        C._row = A_row;

        //calculate size of result matrix
        int32_t C_len = (int64_t)A_col * B_col / B_row ;

        C._col = C_len;

        //caculate size of result matrix

        size_t size_C = C_len * sizeof(stp_data);       

        stp_data *d_C;

        //allocate memory
        hipError_t errC;

        errC = hipMalloc((void **)&d_C, size_C);
        if (errC != hipSuccess) 
        {
            std::cerr << "Error allocating memory for my_semi_tensor_product d_C: " << hipGetErrorString(errC) << std::endl;
        }

        //calculate block size (maximum 1024)
        dim3 threadsPerBlock(1024, 1);

        if(C_len <= Total_Thread)
        {
            //calculate grid size (for large scale)
            dim3 numBlocks((C_len + 1024 - 1 ) / 1024, 1);
            Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(0, A.d_Vec, A._col, B.d_Vec, B._col, d_C, C_len, A_col / B_row);

            //wait for all threads to complete
            hipDeviceSynchronize();          
        }
        else
        {
            int32_t remain_num = C_len; //remaining unassigned columns in C
            int32_t idx_offset = 0;  //thread offset

            while(remain_num)
            {
                //the last  
                if(remain_num <= Total_Thread)
                {
                    //calculate grid size (for large scale) 
                    dim3 numBlocks(( remain_num + 1024 -1 ) / 1024, 1);

                    Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(idx_offset, A.d_Vec, A._col,B.d_Vec, B._col, d_C, C_len, A_col / B_row);
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //calculate thread offset
                    idx_offset += remain_num;
                    remain_num = 0; //exit the loop
                }
                //total thread
                else
                {
                    //calculate grid size (for large scale)
                    dim3 numBlocks(( Total_Thread + 1024 -1 ) / 1024, 1);

                    Matrix_Multipiy_Kernel<<<numBlocks, threadsPerBlock>>>(idx_offset, A.d_Vec, A_col, B.d_Vec, B_col, d_C, C_len, A_col / B_row);
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //calculate thread offset
                    //idx_offset += Total_Thread;
                    hipDeviceSynchronize(); //wait for the kernel to complete
                    //calculate thread offset
                    idx_offset += Total_Thread;
                    remain_num -= Total_Thread; //exit the loop               
                }
            }            
        }

        //free resources
        // if(A_col>20)
        // {
        //     hipFree(A.d_Vec);
        // }
        // if(B_col>20)
        // {
        //     hipFree(B.d_Vec);
        // }


        hipFree(A.d_Vec);
        hipFree(B.d_Vec);
        C.d_Vec = d_C;

        return C;
    }
    else if(B_row % A_col == 0)
    {
        CUDA_DATA temp = my_cuda_Matrix_KR_In(B_row / A_col, A);
        CUDA_DATA C = my_cuda_semi_tensor_product(temp, B);

        return C;
    }
    else
    {
        //error
        std::cout << "Error" << std::endl;
    }
    
}









